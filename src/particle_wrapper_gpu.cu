#include "particle_wrapper_gpu.h"
#include <iostream>

#ifdef __HIPCC__

void particle_wrapper_gpu::setup_cuda_memory() {
    int n = pset.n;
    hipMalloc(&d_positions, n * sizeof(vec2d_t));
    hipMalloc(&d_velocities, n * sizeof(vec2d_t));
    hipMalloc(&d_mass, n * sizeof(scalar_t));
    device_outdated = true;
}

void particle_wrapper_gpu::host_to_device()
{
    int n = pset.n;
    hipMemcpy(d_positions, pset.positions, n * sizeof(vec2d_t), hipMemcpyHostToDevice);
    hipMemcpy(d_velocities, pset.velocities, n * sizeof(vec2d_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mass, pset.mass, n * sizeof(scalar_t), hipMemcpyHostToDevice);
    device_outdated = false;
}

void particle_wrapper_gpu::device_to_host()
{
    int n = pset.n;
    hipMemcpy(pset.positions, d_positions, n * sizeof(vec2d_t), hipMemcpyDeviceToHost);
    hipMemcpy(pset.velocities, d_velocities, n * sizeof(vec2d_t), hipMemcpyDeviceToHost);
    hipMemcpy(pset.mass, d_mass, n * sizeof(scalar_t), hipMemcpyDeviceToHost);
    host_outdated = false;
}

particle_wrapper_gpu::~particle_wrapper_gpu()
{
    hipFree(d_positions);
    hipFree(d_velocities);
    hipFree(d_mass);
}

void particle_wrapper_gpu::do_timestep(simulation_settings_t& settings)
{
    if(this->device_outdated) host_to_device();

    int blockcnt = (pset.n + THREAD_COUNT - 1) / THREAD_COUNT;
    simulation_gpu<<<blockcnt, THREAD_COUNT>>>(pset.n, d_positions, d_velocities, d_mass, settings);
    posupdate_gpu<<<blockcnt, THREAD_COUNT>>>(pset.n, d_positions, d_velocities, settings);
    this->host_outdated = true;
    hipDeviceSynchronize();
    device_to_host();
}

void particle_wrapper_gpu::set_particle_values(int32_t i, const vec2d_t& pos, const vec2d_t& vel, scalar_t mass)
{
    this->device_outdated = true;
    particle_wrapper::set_particle_values(i, pos, vel, mass);
}

vec2d_t particle_wrapper_gpu::get_particle_position(int32_t i)
{
    if(this->host_outdated) device_to_host();
    return particle_wrapper::get_particle_position(i);
}

#endif