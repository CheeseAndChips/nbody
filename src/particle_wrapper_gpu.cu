
#include <hip/hip_runtime.h>
#ifdef USING_CUDA

#include "particle_wrapper_gpu.h"
#include <iostream>

void particle_wrapper_gpu::setup_cuda_memory() {
    int n = pset.n;
    padded_n = n + (n % THREAD_COUNT != 0 ? THREAD_COUNT : 0); 

    cudaMalloc(&d_positions, padded_n * sizeof(vec2d_t));
    cudaMalloc(&d_velocities, padded_n * sizeof(vec2d_t));
    cudaMalloc(&d_mass, padded_n * sizeof(scalar_t));

    cudaMemset(d_mass, 0, padded_n * sizeof(scalar_t)); // ensure that padded particles have mass of 0
}

void particle_wrapper_gpu::host_to_device()
{
    auto n = pset.n;
    cudaMemcpy(d_positions, pset.positions, n * sizeof(vec2d_t), cudaMemcpyHostToDevice);
    cudaMemcpy(d_velocities, pset.velocities, n * sizeof(vec2d_t), cudaMemcpyHostToDevice);
    cudaMemcpy(d_mass, pset.mass, n * sizeof(scalar_t), cudaMemcpyHostToDevice);
    device_outdated = false;
}

void particle_wrapper_gpu::device_to_host()
{
    auto n = pset.n;
    cudaMemcpy(pset.positions, d_positions, n * sizeof(vec2d_t), cudaMemcpyDeviceToHost);
    cudaMemcpy(pset.velocities, d_velocities, n * sizeof(vec2d_t), cudaMemcpyDeviceToHost);
    cudaMemcpy(pset.mass, d_mass, n * sizeof(scalar_t), cudaMemcpyDeviceToHost);
    host_outdated = false;
}

particle_wrapper_gpu::~particle_wrapper_gpu()
{
    cudaFree(d_positions);
    cudaFree(d_velocities);
    cudaFree(d_mass);
}

void particle_wrapper_gpu::do_timestep(simulation_settings_t& settings)
{
    wait_for_lock();
    this->ongoing_calculation = true;    
    if(this->device_outdated) host_to_device();

    int blockcnt = padded_n / THREAD_COUNT;
    simulation_gpu<<<blockcnt, THREAD_COUNT>>>(padded_n, d_positions, d_velocities, d_mass, settings);
    posupdate_gpu<<<blockcnt, THREAD_COUNT>>>(padded_n, d_positions, d_velocities, settings);
    this->host_outdated = true;
}

void particle_wrapper_gpu::set_particle_values(int32_t i, const vec2d_t& pos, const vec2d_t& vel, scalar_t mass)
{
    if(ongoing_calculation) {
        PyErr_SetString(PyExc_BlockingIOError, "Trying to change particle values with ongoing calculations");
        return;
    }

    if(this->host_outdated) device_to_host();
    particle_wrapper::set_particle_values(i, pos, vel, mass);
    this->device_outdated = true;
}

vec2d_t particle_wrapper_gpu::get_particle_position(int32_t i)
{
    if(!this->ongoing_calculation && this->host_outdated) device_to_host();
    return particle_wrapper::get_particle_position(i);
}

void particle_wrapper_gpu::wait_for_lock()
{
    if(ongoing_calculation) {
        cudaDeviceSynchronize();
        ongoing_calculation = false;
        device_to_host();
    }
}

#endif